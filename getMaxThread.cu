#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>


int getThreadNum()
{
    hipDeviceProp_t prop;
    int count;
    int dev;
    hipGetDeviceCount(&count);
    printf("gpu num %d\n", count);
    hipGetDeviceProperties(&prop, 0);
    printf("max thread num: %d\n", prop.maxThreadsPerBlock);
    printf("max grid dimensions: %d, %d, %d)\n",
    prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    return prop.maxThreadsPerBlock;
}

int main(void)
{
    getThreadNum();
    return 0;
}
