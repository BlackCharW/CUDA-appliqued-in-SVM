#include <iostream>
#include <hip/hip_runtime.h>


using namespace std;

__global__ void vec_add(int *a,int *b)
{
    int i = blockIdx.x;
    a[i] +=b [i];
}

int main()
{
    const int N = 100;
    int *a, *b;
    hipMallocManaged(&a,N*sizeof(int));
    hipMallocManaged(&b,N*sizeof(int));

    for(int i = 0;i<N;i++)
    {
        a[i] = i;
        b[i] = 2*i;

    }

    vec_add<<<N,1>>>(a,b);
    hipDeviceSynchronize();
    for(int i=0;i<N;i++)
    {
        cout << a[i] << endl;
    }
}